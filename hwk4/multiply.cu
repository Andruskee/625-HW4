//
// mutliply.cu - example matrix multiplication in CUDA
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

//
// gpu_matrix_multiply
//
// grid and block configuration:
//
//   dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
//   dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//

__global__ void gpu_matrix_multiply(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int h = 0; h < n; h++) 
        {
            sum += a[row * n + h] * b[h * k + col];
        }
        c[row * k + col] = sum;
    }
} 

//
// cpu_matrix_multiply - to verify results
//
void cpu_matrix_multiply(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
        for (int j = 0; j < k; ++j) {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
                tmp += h_a[i * n + h] * h_b[h * k + j];
            h_result[i * k + j] = tmp;
        }
}

//
// main
//
int main(int argc, char const *argv[])
{
    int m, n, k;

    srand(123);

    if (argc > 3) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        k = atoi(argv[3]);
    } else {
        m = 400;
        n = 500;
        k = 600;
    }

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k, hipHostMallocDefault);

    // randomly init A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 512;
        }
    }

    // randomly init B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 512;
        }
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // record start execution time of GPU version
    hipEventRecord(start, 0);

    // allocate memory on CUDA device 
    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    // execute kernel 
    gpu_matrix_multiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);    

    // copy results from device to host 
    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // record end time working on CUDA
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapsed
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Elapsed time for A %dx%d times B %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);

    // compute using CPU
    hipEventRecord(start, 0);

    cpu_matrix_multiply(h_a, h_b, h_cc, m, n, k);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Elapsed time on CPU: %f ms.\n\n", cpu_elapsed_time_ms);

    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            //printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_cc[i*k + j], i, j, h_c[i*k + j]);
            if(h_cc[i*k + j] != h_c[i*k + j])
            {
                all_ok = 0;
            }
        }
        //printf("\n");
    }

    // roughly compute speedup
    if(all_ok)
    {
        printf("Identical results on GPU and CPU. Speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("Results don't match.\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}
